#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__device__ double calcDistance(const Point* p1, const Point* p2, double* t) {
    double x1 = ((p1->x2 - p1->x1) / 2) * sin((*t) * M_PI / 2) + ((p1->x2 + p1->x1) / 2);
    double y1 = p1->a * x1 + p1->b;

    double x2 = ((p2->x2 - p2->x1) / 2) * sin((*t) * M_PI / 2) + ((p2->x2 + p2->x1) / 2);
    double y2 = p2->a * x2 + p2->b;

    double distance = sqrt(pow(x2 - x1, 2) + pow(y2 - y1, 2));

    return distance;
}

__global__ void checkProximityCriteria(int* count, const Point *points, double *tValues, const int tCount,const int N,const int K, const double D, double* distances)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // point idx

    if (idx < tCount)
    {       
        double* currentTValue = &(tValues[idx]);
        for (int i = 0; i < N; i++) {
            const Point currentPoint = points[i];
            for(int j = 0; j < N && j != i; j++) {
                const Point otherPoint = points[j];
                double distance = calcDistance(&currentPoint, &otherPoint, currentTValue);

                if (distance <= D) {
                    atomicAdd(count, 1);
                    if ((*count) >= K) {
                        break;
                    }
                }
                distances[idx * N * (N - 1) + i * (N - 1) + j] = distance;
            }
        }
    }
}


void computeOnGPU(int *N, int *K, double *D, int *tCountSize, double *myTValues, Point *points) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    int blocksPerGrid = ((*tCountSize) + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int threadsPerBlock = BLOCK_SIZE;

    Point *dPoints;         // points for device
    double *dTValues;       // tValues for device
    double *dDistances;     // distances array for device

    err = hipMalloc(&dPoints, (*N) * sizeof(Point));
    if (err != hipSuccess){
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc(&dTValues, (*tCountSize) * sizeof(double));
    if (err != hipSuccess){
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc(&dDistances, (*tCountSize) * (*N) * (*N - 1) * sizeof(double));
    if (err != hipSuccess){
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(dPoints, points, (*N) * sizeof(Point), hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(dTValues, myTValues, (*tCountSize) * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int count = 0;
    checkProximityCriteria<<<blocksPerGrid, threadsPerBlock>>>(&count, dPoints, dTValues, *tCountSize, *N, *K, *D, dDistances);

    double *distances = (double*)malloc((*tCountSize) * (*N) * (*N - 1) * sizeof(double));
    if (distances == NULL){
        fprintf(stderr, "Error allocating host memory!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(distances, dDistances, (*tCountSize) * (*N) * (*N - 1) * sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess){
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Print the distances
    for (int idx = 0; idx < (*tCountSize); idx++) {
        for (int i = 0; i < (*N); i++) {
            for (int j = 0; j < (*N - 1); j++) {
                printf("%d) Point %d and point %d - distance %lf\n", idx, i, j, distances[idx * (*N) * (*N - 1) + i * (*N - 1) + j]);
            }
        }
    }

    hipFree(dPoints);
    hipFree(dTValues);
    hipFree(dDistances);
    free(distances);
}
