#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__device__ double calcDistance(const Point* p1, const Point* p2, double* t) {
    double x1 = ((p1->x2 - p1->x1) / 2) * sin((*t) * M_PI) + ((p1->x2 + p1->x1) / 2);
    double y1 = p1->a * x1 + p1->b;

    double x2 = ((p2->x2 - p2->x1) / 2) * sin((*t) * M_PI) + ((p2->x2 + p2->x1) / 2);
    double y2 = p2->a * x2 + p2->b;

    double distance = sqrt(pow(x2 - x1, 2) + pow(y2 - y1, 2));

    return distance;
}

__global__ void checkProximityCriteria(int* count, Point *points, double *tValues, const int tCount, const int N, const int K, const double D, int **results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // point idx

    if (idx < tCount) {       
        double t = tValues[idx];

        for (int i = 0; i < N; i++) {
            for (int j = 0; j < N && i != j; j++) {
                double distance = calcDistance(&points[i], &points[j], &t);
                
                printf("t = %d - point %d and point %d - distance %lf\n",idx, i,j,distance);
                if (distance <= D) {
                    int currentCount = atomicAdd(count, 1);
                    if (currentCount < K) {
                        results[currentCount][idx] = points[i].id;
                    }
                }

                if (*count >= K) {
                    break;
                }
            }

            if (*count >= K) {
                break;
            }
        }
    }
}


void computeOnGPU(int *count, int *N, int *K, double *D, int *tCountSize, double *myTValues, Point *points, int **results) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    int threadPerBlock = *tCountSize < BLOCK_SIZE ? *tCountSize : BLOCK_SIZE;
    int blocksPerGrid = ((*tCountSize) * (*N)) / threadPerBlock < 1 ? 1 : round(((*tCountSize) * (*N)) / threadPerBlock);
    int* d_count = NULL;
    Point* d_points = NULL;
    double* d_tValues = NULL;
    int ** d_results = NULL;

    // Allocate the device 
    err = hipMalloc((void **)&d_count, sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device count (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_points, (*N) * sizeof(Point));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device points (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_tValues, (*tCountSize) * sizeof(double));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device tValues (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_results, (*tCountSize) * sizeof(int*));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device results (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    int** d_results_host = (int**)malloc((*N) * sizeof(int*));
    for (int i = 0; i < *N; i++) {
        err = hipMalloc((void **)&d_results_host[i], (*tCountSize) * sizeof(int));
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to allocate device results row %d (error code %s)!\n", i, hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    // Copy from host to device
    err = hipMemcpy(d_points, points, (*N) * sizeof(Point), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy points from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_tValues, myTValues, (*tCountSize) * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy tValues from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_count, count, sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy count from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    for (int i = 0; i < *N; i++) {
        err = hipMemcpy(d_results_host[i], results[i], (*tCountSize) * sizeof(int), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to copy results row %d from host to device (error code %s)!\n", i, hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }
    err = hipMemcpy(d_results, d_results_host, (*N) * sizeof(int*), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy results array from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Launch the proximity criteria check on the GPU
    checkProximityCriteria<<<blocksPerGrid, threadPerBlock>>>(d_count, d_points, d_tValues, *tCountSize, *N, *K, *D, d_results);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch checkProximityCriteria kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy from device to host
    err = hipMemcpy(count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy count from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    for (int i = 0; i < *N; i++) {
        err = hipMemcpy(results[i], d_results_host[i], (*tCountSize) * sizeof(int), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to copy results row %d from device to host (error code %s)!\n", i, hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    // Free device memory
    hipFree(d_count);
    hipFree(d_points);
    hipFree(d_tValues);
    for (int i = 0; i < *tCountSize; i++) {
        hipFree(d_results_host[i]);
    }
    hipFree(d_results);
}
