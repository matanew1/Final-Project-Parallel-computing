#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__device__ double calcDistance(const Point* p1, const Point* p2, double* t) {
    double x1 = ((p1->x2 - p1->x1) / 2) * sin((*t) * M_PI) + ((p1->x2 + p1->x1) / 2);
    double y1 = p1->a * x1 + p1->b;

    double x2 = ((p2->x2 - p2->x1) / 2) * sin((*t) * M_PI) + ((p2->x2 + p2->x1) / 2);
    double y2 = p2->a * x2 + p2->b;

    double distance = sqrt(pow(x2 - x1, 2) + pow(y2 - y1, 2));

    return distance;
}

__global__ void checkProximityCriteria(int* count, Point *points, double *tValues, const int tCount,const int N,const int K, const double D){
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // point idx

    if (idx < tCount)
    {       
        double t = tValues[idx];

        for (int i = 0; i < N; i++) {
            for(int j = 0; j < N && i != j; j++) {
                double distance = calcDistance(&points[i], &points[j], &t);

                // printf("t = %d point %d and point %d - distance = %lf\n",idx, i, j, distance);
                // printf("\tpoint %d (x1=%.2lf x2=%.2lf a=%.2lf b=%.2lf) and point %d (x1=%.2lf x2=%.2lf a=%.2lf b=%.2lf)\n",
                // i,points[i].x1, points[i].x2, points[i].a, points[i].b,
                // j,points[j].x1, points[j].x2, points[j].a, points[j].b);
                if( distance <= D ) {
                    atomicAdd(count, 1);
                }
                if (*count >= K) {
                    break;
                }
            }         
        }
    }
}

void computeOnGPU(int *count, int *N, int *K, double *D, int *tCountSize, double *myTValues, Point *points) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    int threadPerBlock = *tCountSize < BLOCK_SIZE ? *tCountSize : BLOCK_SIZE;
    int blocksPerGrid = ((*tCountSize) * (*N)) / threadPerBlock < 1 ? 1 : round(((*tCountSize) * (*N)) / threadPerBlock);
    int* d_count = NULL;
    Point* d_points = NULL;
    double* d_tValues = NULL;

    // Allocate the device 
    err = hipMalloc((void **)&d_count, sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device count (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_points, (*N) * sizeof(Point));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device points (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_tValues, (*tCountSize) * sizeof(double));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device tValues (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy from host to device
    err = hipMemcpy(d_points, points, (*N) * sizeof(Point), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy points from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_tValues, myTValues, (*tCountSize) * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy tValues from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_count, count, sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy count from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the proximity criteria check on the GPU
    checkProximityCriteria<<<blocksPerGrid, threadPerBlock>>>(d_count, d_points, d_tValues, *tCountSize, *N, *K, *D);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch checkProximityCriteria kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy from device to host
    err = hipMemcpy(count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy count from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device memory
    hipFree(d_count);
    hipFree(d_points);
    hipFree(d_tValues);
}