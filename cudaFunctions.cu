#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__device__ double calcDistance(const Point *p1, const Point *p2, double *t)
{
    double x1 = ((p1->x2 - p1->x1) / 2) * sin((*t) * M_PI / 2) + ((p1->x2 + p1->x1) / 2);
    double y1 = p1->a * x1 + p1->b;

    double x2 = ((p2->x2 - p2->x1) / 2) * sin((*t) * M_PI / 2) + ((p2->x2 + p2->x1) / 2);
    double y2 = p2->a * x2 + p2->b;

    double distance = sqrt(pow(x2 - x1, 2) + pow(y2 - y1, 2));

    return distance;
}

__global__ void checkProximityCriteria(Point *points, double *tValues, const int tCount, const int N, const int K, const double D, int *results)
{
    int count = 0;
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // point idx
    if (idx < tCount)
    {
        double t = tValues[idx];

        for (int i = 0; i < N; i++)
        {
            for (int j = 0; j < N && i != j; j++)
            {
                double distance = calcDistance(&points[i], &points[j], &t);
                
                printf("T = %d || count = %d\n",idx, count);
                if (distance <= D && distance > 0)
                {
                    count++;
                    if (count == K) {                        
                        for (int i = 0; i < tCount; i++) {                      
                            if ( i == idx ) {
                                for (int j = 0; j < CONSTRAINTS; j++) { 
                                    if (results[i * tCount + j] == -1) {
                                        printf("t = %d || with point %d || res_index = %d\n",idx,points[i].id, i * tCount + j);
                                        atomicExch(&results[i * tCount + j], points[i].id); 
                                        return;  
                                    }
                                }
                            }
                        }                                                                                       
                    }                          
                }
            }
        }
    }
}

void computeOnGPU(int *N, int *K, double *D, int *tCountSize, double *myTValues, Point *points, int *results)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    int threadPerBlock = *tCountSize < BLOCK_SIZE ? *tCountSize : BLOCK_SIZE;
    int blocksPerGrid = 1;

    Point *d_points = NULL;
    double *d_tValues = NULL;
    int *d_results = NULL;

    // Allocate the device
    err = hipMalloc((void **)&d_points, (*N) * sizeof(Point));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device points (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_tValues, (*tCountSize) * sizeof(double));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device tValues (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_results, (CONSTRAINTS) * (*tCountSize) * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device results (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy from host to device
    err = hipMemcpy(d_points, points, (*N) * sizeof(Point), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy points from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_tValues, myTValues, (*tCountSize) * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy tValues from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_results, results, (CONSTRAINTS) * (*tCountSize) * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy results from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the proximity criteria check on the GPU
    checkProximityCriteria<<<blocksPerGrid, threadPerBlock>>>(d_points, d_tValues, *tCountSize, *N, *K, *D, d_results);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch checkProximityCriteria kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy from device to host
    err = hipMemcpy(results, d_results, (CONSTRAINTS) * (*tCountSize) * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy results from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < *tCountSize; i++)
    {
        printf("current t %d\n", i);
        for (int j = 0; j < CONSTRAINTS; j++)
        {
            printf("\tp[%d] = %d ", j, results[i * (CONSTRAINTS) + j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_points);
    hipFree(d_tValues);
    hipFree(d_results);
}
