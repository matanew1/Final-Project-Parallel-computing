#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

/**
 * Calculate the distance between two points.
 * @param p1 Struct that points to one point.
 * @param p2 Struct that points to the second point.
 * @param t The current t-value.
 * @return The distance between the two points.
 */
__device__ double calcDistance(const Point p1, const Point p2, double t)
{
    double x1 = ((p1.x2 - p1.x1) / 2) * __sinf(t * M_PI / 2) + ((p1.x2 + p1.x1) / 2);
    double y1 = p1.a * x1 + p1.b;

    double x2 = ((p2.x2 - p2.x1) / 2) * __sinf(t * M_PI / 2) + (p2.x2 + p2.x1) / 2;
    double y2 = p2.a * x2 + p2.b;

    double dx = x2 - x1;
    double dy = y2 - y1;

    return sqrt(dx * dx + dy * dy);
}

/**
 * Update proximity points in an atomic manner.
 * @param startingIndex The current t-value in the round.
 * @param results Array of results.
 * @param pointId A point that satisfies the condition.
 */
__device__ void updateProximitePoints(int startingIndex, int *resutls, int pointId)
{
    for (int i = 0; i < CONSTRAINTS; i++)
    {
        int index = startingIndex * CONSTRAINTS + i;
        int currentVal = resutls[index];
        if (currentVal == -1)
        {
            int expected = -1;
            int desired = pointId;

            if (atomicCAS(&resutls[index], expected, desired) == expected) /*Fill the resutls[index] in atomic way*/
            {
                return;
            }
        }
    }
}

/**
 * Check proximity of points on the GPU.
 * @param d_points Array of all N points.
 * @param N Number of points.
 * @param tValue Current t-value.
 * @param D Max distance to check.
 * @param d_results Array of results.
 * @param K Need at least K points that fulfill the condition of Proximity Criteria.
 * @param tIndex The current t in the for loop.
 */
__global__ void checkProximity(Point *d_points, int N, double tValue, double D, int *d_resutls, double K, int tIndex)
{
    int pid = blockDim.x * blockIdx.x + threadIdx.x;

    if (pid >= N || d_resutls[tIndex * CONSTRAINTS + CONSTRAINTS - 1] != -1)
    {
        return;
    }

    int counter = 0;
    int currentPoint = d_points[pid];

    for (int i = 0; i < N && counter < K; i++)
    {
        int checkedPoint = d_points[i];
        if (checkedPoint.id != currentPoint.id && calcDistance(currentPoint, checkedPoint, tValue) < D)
        {
            counter++;
        }
    }

    if (counter == K)
    {
        updateProximitePoints(tIndex, d_resutls, currentPoint.id);
    }
}


/**
 * Allocate memory on the GPU.
 * @param ptr Pointer to the memory to be allocated.
 * @param size Size of the memory to allocate (in bytes).
 */
void allocateMemDevice(void **ptr, size_t size)
{
    hipError_t err = hipMalloc(ptr, size); 
    if (err != hipSuccess)
    {
        fprintf(stderr, "Cannot to allocate memory on device. -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * Copy memory between host and device.
 * @param dest Pointer to the destination in device/host memory.
 * @param src Pointer to the source in host/device memory.
 * @param size How much data needs to be copied (in bytes).
 * @param direction Which direction to copy the memory (host->device) or (device->host).
 */
void copyMemory(void *dest, void *src, size_t size, hipMemcpyKind direction)
{
    /*Copy mem from device to host OR host to device depending on the direction*/
    hipError_t err = hipMemcpy(dest, src, size, direction);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data. -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


/**
 * Perform computation on the GPU.
 * @param N Number of points.
 * @param K Need at least K points that fulfill the condition of Proximity Criteria.
 * @param D Max distance to check.
 * @param tCount Number of t values.
 * @param tValues Array of t values.
 * @param points Array of points.
 * @param results Array of results.
 * @return 0 if the computation is successful.
 */
int computeOnGPU(int N, int K, double D, int tCount, double *tValues, Point *points, int *results)
{
    hipError_t err = hipSuccess;

    int threadPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (N + threadPerBlock - 1) / threadPerBlock;
    Point *d_points = NULL;
    int *d_resutls = NULL;

    /*Allocating mem on gpu section*/
    allocateMemDevice((void **)&d_resutls, CONSTRAINTS * tCount * sizeof(int)); 
    allocateMemDevice((void **)&d_points, N * sizeof(Point));
    /*End allocate mem*/

    /*Copy mem to Device section*/
    copyMemory(d_points, points, N * sizeof(Point), hipMemcpyHostToDevice);
    copyMemory(d_resutls, results, tCount * CONSTRAINTS * sizeof(int), hipMemcpyHostToDevice);
    /*End copy mem to Device*/

    /*for each tvalue we will send it to GPU to compute the data and save it on resutls array*/
    for (int i = 0; i < tCount; i++)
    {
        checkProximity<<<blocksPerGrid, threadPerBlock>>>(d_points, N, tValues[i], D, d_resutls, K, i);
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to lanch checkProximity kernel. -%s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    err = hipMemcpy(results, d_resutls, tCount * CONSTRAINTS * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data. -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data. -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (hipFree(d_points) != hipSuccess)
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if (hipFree(d_resutls) != hipSuccess)
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return 0;
}
