#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

/**
 * Calculate the distance between two points at a given t value.
 *
 * @param p1  Pointer to the first point
 * @param p2  Pointer to the second point
 * @param t   Pointer to the t value
 * @return    The calculated distance
 */
__device__ double calcDistance(const Point *p1, const Point *p2, double *t)
{
    // Calculate the x-coordinate of the first point at a given value of t.
    // x1 = ((x2 - x1) / 2) * sin(t * π / 2) + ((x2 + x1) / 2);
    double x1 = ((p1->x2 - p1->x1) / 2) * __fsin((*t) * M_PI / 2) + ((p1->x2 + p1->x1) / 2);

    // Calculate the y-coordinate of the first point based on the equation of a line.
    // y1 = a * x1 + b;
    double y1 = p1->a * x1 + p1->b;

    // Calculate the x-coordinate of the second point at a given value of t.
    // x2 = ((x2 - x1) / 2) * sin(t * π / 2) + ((x2 + x1) / 2);
    double x2 = ((p2->x2 - p2->x1) / 2) * __fsin((*t) * M_PI / 2) + ((p2->x2 + p2->x1) / 2);

    // Calculate the y-coordinate of the second point based on the equation of a line.
    // y2 = a * x2 + b;
    double y2 = p2->a * x2 + p2->b;

    // Calculate the distance between the two points using the Euclidean distance formula.
    // distance = sqrt((x2 - x1)^2 + (y2 - y1)^2);
    double distance = sqrt(pow(x2 - x1, 2) + pow(y2 - y1, 2));

    double distanceSquared = dx * dx + dy * dy; // Avoiding square root for Euclidean distance check

    return distanceSquared <= D * D; // Compare with squared threshold
}

__global__ void checkProximityCriteria(Point *points, double *tValues, const int tCount, const int N, const int K, const double D, int *results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= tCount)
        return;

    double t = tValues[idx];
    __shared__ int sharedResults[BLOCK_SIZE * CONSTRAINTS];

    for (int j = threadIdx.x; j < CONSTRAINTS; j += blockDim.x) {
        sharedResults[threadIdx.x * CONSTRAINTS + j] = -1;
    }

    __syncthreads();

    int pointId = -1;

    for (int i = threadIdx.x; i < N; i += blockDim.x) {
        int count = 0;

        for (int j = 0; j < N; j++) {
            if (i != j && isProximityCriteriaMet(&points[i], &points[j], &t, D)) {
                count++;

                if (count == K) {
                    pointId = points[i].id;
                    break;
                }
            }
        }

        if (pointId != -1) {
            break;
        }
    }

    // Update sharedResults
    for (int j = threadIdx.x; j < CONSTRAINTS; j += blockDim.x) {
        if (pointId != -1 && sharedResults[j] == -1) {
            sharedResults[j] = pointId;
        }
    }

    __syncthreads();

    // Have one thread per block write sharedResults to global results
    if (threadIdx.x == 0) {
        for (int j = 0; j < CONSTRAINTS; j++) {
            int targetIndex = idx * CONSTRAINTS + j;
            results[targetIndex] = sharedResults[j];
        }
    }
}

void computeOnGPU(int N, int K, double D, int tCountSize, double *myTValues, Point *points, int *results) {
    hipError_t err = hipSuccess;
    int threadPerBlock = min(BLOCK_SIZE, tCountSize);
    int blocksPerGrid = (tCountSize + threadPerBlock - 1) / threadPerBlock;

    Point *d_points = nullptr;
    double *d_tValues = nullptr;
    int *d_results = nullptr;

    allocateDeviceMemory((void **)&d_points, N * sizeof(Point));
    allocateDeviceMemory((void **)&d_tValues, tCountSize * sizeof(double));
    allocateDeviceMemory((void **)&d_results, CONSTRAINTS * tCountSize * sizeof(int));

    copyHostToDevice(d_points, points, N * sizeof(Point), hipMemcpyHostToDevice);
    copyHostToDevice(d_tValues, myTValues, tCountSize * sizeof(double), hipMemcpyHostToDevice);
    copyHostToDevice(d_results, results, CONSTRAINTS * tCountSize * sizeof(int), hipMemcpyHostToDevice);

    // Allocate device memory for points, tValues, and results.
    allocateDeviceMemory((void **)&d_points, (*N) * sizeof(Point));
    allocateDeviceMemory((void **)&d_tValues, (*tCountSize) * sizeof(double));
    allocateDeviceMemory((void **)&d_results, CONSTRAINTS * (*tCountSize) * sizeof(int));
    printf("Allocated device memory for points, tValues, and results\n");

    // Copy points, tValues, and results from the host to the device.
    copyHostToDevice(d_points, points, (*N) * sizeof(Point), hipMemcpyHostToDevice);
    copyHostToDevice(d_tValues, myTValues, (*tCountSize) * sizeof(double), hipMemcpyHostToDevice);
    copyHostToDevice(d_results, results, CONSTRAINTS * (*tCountSize) * sizeof(int), hipMemcpyHostToDevice);
    printf("Copy host memory to device for points, tValues, and results\n");

    // Launch the checkProximityCriteria kernel on the device.
    checkProximityCriteria<<<blocksPerGrid, threadPerBlock>>>(d_points, d_tValues, *tCountSize, *N, *K, *D, d_results);

    // Check if there was an error launching the kernel.
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipDeviceSynchronize();
    copyDeviceToHost(results, d_results, CONSTRAINTS * tCountSize * sizeof(int), hipMemcpyDeviceToHost);

    freeDeviceMemory(d_points);
    freeDeviceMemory(d_tValues);
    freeDeviceMemory(d_results);
}
