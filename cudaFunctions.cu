#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__device__ double calcDistance(const Point *p1, const Point *p2, double *t)
{
    double x1 = ((p1->x2 - p1->x1) / 2) * sin((*t) * M_PI / 2) + ((p1->x2 + p1->x1) / 2);
    double y1 = p1->a * x1 + p1->b;

    double x2 = ((p2->x2 - p2->x1) / 2) * sin((*t) * M_PI / 2) + ((p2->x2 + p2->x1) / 2);
    double y2 = p2->a * x2 + p2->b;

    double distance = sqrt(pow(x2 - x1, 2) + pow(y2 - y1, 2));

    return distance;
}

__device__ bool isProximityCriteriaMet(const Point *p1, const Point *p2, double *t, double D)
{
    double distance = calcDistance(p1, p2, t);
    // printf("t = %.3lf || point %d, point %d || distance = %.3lf\n",*t,p1->id,p2->id,distance);
    return distance <= D;
}

__device__ void updateResults(int idx, int *results, int proximityPointId)
{
    for (int j = 0; j < CONSTRAINTS; j++)
    {
        if (results[idx * CONSTRAINTS + j] == -1)
        {
            atomicExch(&results[idx * CONSTRAINTS + j], proximityPointId);
            // printf("res_p%d || idx_t = %d || with point %d || res_index = %d\n",
            // j,idx,proximityPointId, idx * CONSTRAINTS + j);
            break;
        }
    }
}

__global__ void checkProximityCriteria(Point *points, double *tValues, const int tCount, const int N, const int K, const double D, int *results)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // point idx

    if (idx >= tCount) return; // specific t

    double t = tValues[idx];
    // printf("t = %d value = %.3lf\n",idx, t);
    int count = 0;
    int finish = 0;

    for (int i = 0; i < N; i++)
    {
        count = 0;
        finish = 0;
        for (int j = 0; j < N; j++)
        {
            if (finish == 1) break;
            if (i != j && isProximityCriteriaMet(&points[i], &points[j], &t, D))
            {
                count++;
                if (count == K)
                {
                    int proximityPointId = points[i].id;
                    updateResults(idx, results, proximityPointId);
                    finish = 1;
                }
            }
        }
    }
}

void computeOnGPU(int *N, int *K, double *D, int *tCountSize, double *myTValues, Point *points, int *results)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // TODO: need to fix this section !!!/////////////////////////////
    int threadPerBlock = min(BLOCK_SIZE, *tCountSize);              //
    int blocksPerGrid = *tCountSize / BLOCK_SIZE < 1 ? 1 : *tCountSize / BLOCK_SIZE; //
    // printf("*tCountSize = %d threadPerBlock=%d blocksPerGrid=%d\n", //
    //  *tCountSize,threadPerBlock,blocksPerGrid);                      //
    //////////////////////////////////////////////////////////////////

    Point *d_points = NULL;
    double *d_tValues = NULL;
    int *d_results = NULL;

    // Allocate the device
    err = hipMalloc((void **)&d_points, (*N) * sizeof(Point));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device points (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_tValues, (*tCountSize) * sizeof(double));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device tValues (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc((void **)&d_results, (CONSTRAINTS) * (*tCountSize) * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device results (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy from host to device
    err = hipMemcpy(d_points, points, (*N) * sizeof(Point), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy points from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_tValues, myTValues, (*tCountSize) * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy tValues from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_results, results, (CONSTRAINTS) * (*tCountSize) * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy results from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the proximity criteria check on the GPU
    checkProximityCriteria<<<blocksPerGrid, threadPerBlock>>>(d_points, d_tValues, *tCountSize, *N, *K, *D, d_results);
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch checkProximityCriteria kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy from device to host
    err = hipMemcpy(results, d_results, (CONSTRAINTS) * (*tCountSize) * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy results from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    for (int i = 0; i < *tCountSize; i++)
    {
        printf("current t %d\n", i);
        for (int j = 0; j < CONSTRAINTS; j++)
        {
            printf("\tp[%d] = %d ", j, results[i * (CONSTRAINTS) + j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_points);
    hipFree(d_tValues);
    hipFree(d_results);
}
