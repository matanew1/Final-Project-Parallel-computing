#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

__device__ double calcDistance(const Point* p1, const Point* p2, double* t) {
    double x1 = ((p1->x2 - p1->x1) / 2) * sin((*t) * M_PI / 2) + ((p1->x2 + p1->x1) / 2);
    double y1 = p1->a * x1 + p1->b;

    double x2 = ((p2->x2 - p2->x1) / 2) * sin((*t) * M_PI / 2) + ((p2->x2 + p2->x1) / 2);
    double y2 = p2->a * x2 + p2->b;

    double distance = sqrt(pow(x2 - x1, 2) + pow(y2 - y1, 2));

    return distance;
}

__global__ void checkProximityCriteria(int* count, const Point *points, double *tValues, const int tCount,const int N,const int K, const double D){
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // point idx

    if (idx < tCount)
    {
        
        double* t = &(tValues[idx]);
        for (int i = 0; i < N; i++) {
            for(int j = 0; j < N && j != i; j++) {
                double distance = calcDistance(&points[i], &points[j], t);

                if (distance <= D) {
                    (*count)++;
                    if ((*count) >= K) {
                        break;
                    }
                }  
            }         
        }
    }
}


void computeOnGPU(int *N, int *K, double *D, int *tCountSize, double *myTValues, Point *points) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    int blocksPerGrid = ((*tCountSize) + (*N)) / BLOCK_SIZE < 1 ? 1 : ((*tCountSize) + (*N)) / BLOCK_SIZE < 1; 
    int threadsPerBlock = BLOCK_SIZE;

    Point *dPoints;     // point for device
    double *dTValues;   // tValues for device    

    err = hipMalloc(&dPoints, (*N) * sizeof(Point));
    if (err != hipSuccess){
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc(&dTValues, (*tCountSize) * sizeof(double));
    if (err != hipSuccess){
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(dPoints, points, (*N) * sizeof(Point), hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(dTValues, myTValues, (*tCountSize) * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Error in line %d (error code %s)!\n", __LINE__, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int count = 0;

    checkProximityCriteria<<<blocksPerGrid, threadsPerBlock>>>(&count, dPoints, dTValues, *tCountSize, *N, *K, *D);
    printf("Count: %d\n",count);
    
    // free device allocation
    hipFree(dPoints);
    hipFree(dTValues);
}
