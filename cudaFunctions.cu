#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"

/**
 * Calculate the distance between two points.
 * @param p1 Struct that points to one point.
 * @param p2 Struct that points to the second point.
 * @param t The current t-value.
 * @return The distance between the two points.
 */
__device__ double calcDistance(const Point p1, const Point p2, double t)
{
    double x1 = ((p1.x2 - p1.x1) / 2) * __sinf(t * M_PI / 2) + ((p1.x2 + p1.x1) / 2);
    double y1 = p1.a * x1 + p1.b;

    double x2 = ((p2.x2 - p2.x1) / 2) * __sinf(t * M_PI / 2) + (p2.x2 + p2.x1) / 2;
    double y2 = p2.a * x2 + p2.b;

    double dx = x2 - x1;
    double dy = y2 - y1;

    return sqrt(dx * dx + dy * dy);
}

/**
 * Update proximity points in an atomic manner.
 * @param tIdx The current t-value in the round.
 * @param results Array of results.
 * @param pointId A point that satisfies the condition.
 */
__device__ void updateResults(int tIdx, int *results, int pointId)
{
    // Loop through each constraint
    for (int i = 0; i < CONSTRAINTS; i++)
    {
        // Calculate the index in the results array for the current tIdx and constraint
        int index = tIdx * CONSTRAINTS + i;

        // Retrieve the current value stored at the calculated index
        int currentVal = results[index];

        // Check if the current value is -1 (indicating an unset value)
        if (currentVal == -1)
        {
            // Attempt to atomically update the value at the calculated index
            // If the current value is still -1, the swap is performed and returns true
            if (atomicCAS(&results[index], currentVal, pointId) == currentVal)
            {
                // If the swap was successful (i.e., the current value was still -1),
                // exit the loop and the function
                return;
            }
        }
    }
}

/**
 * Check proximity of points on the GPU.
 * @param d_points Array of all N points.
 * @param N Number of points.
 * @param tValue Current t-value.
 * @param D Max distance to check.
 * @param d_results Array of results.
 * @param K Need at least K points that fulfill the condition of Proximity Criteria.
 * @param tIdx The current t in the for loop.
 */
__global__ void checkProximity(Point *d_points, int N, double tValue, double D, int *d_results, double K, int tIdx)
{
    // Calculate the unique thread ID within the grid
    int pid = blockDim.x * blockIdx.x + threadIdx.x;

    // Initialize a counter to track the number of nearby points
    int counter = 0;

    // Check if the thread ID is within the valid range of points
    if (pid < N)
    {
        // Loop through all points to check proximity
        for (int i = 0; i < N; i++)
        {
            // Check if the proximity results have been flagged as complete
            if (atomicAdd(&d_results[tIdx * CONSTRAINTS + CONSTRAINTS - 1], 0) != -1)
                return; // If the results are complete, exit the function immediately

            // Compare the ID of the current point and the checked point
            Point current = d_points[pid];
            Point checked = d_points[i];

            if (checked.id != current.id && calcDistance(current, checked, tValue) < D)
            {
                // Increment the counter when the condition is met
                counter++;

                // Check if the required number of nearby points have been found
                if (counter == K)
                {
                    // Update the results array with the current point's ID
                    updateResults(tIdx, d_results, current.id);
                    // Exit the loop since the required condition has been satisfied
                    break;
                }
            }
        }
    }
}

/**
 * Allocate memory on the GPU.
 * @param ptr Pointer to the memory to be allocated.
 * @param size Size of the memory to allocate (in bytes).
 */
void allocateMemDevice(void **ptr, size_t size)
{
    hipError_t err = hipMalloc(ptr, size); 
    if (err != hipSuccess)
    {
        fprintf(stderr, "Cannot to allocate memory on device. -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * Copy memory between host and device.
 * @param dest Pointer to the destination in device/host memory.
 * @param src Pointer to the source in host/device memory.
 * @param size How much data needs to be copied (in bytes).
 * @param direction Which direction to copy the memory (host->device) or (device->host).
 */
void copyMemory(void *dest, void *src, size_t size, hipMemcpyKind direction)
{
    /*Copy mem from device to host OR host to device depending on the direction*/
    hipError_t err = hipMemcpy(dest, src, size, direction);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data. -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


/**
 * Perform computation on the GPU.
 * @param N Number of points.
 * @param K Need at least K points that fulfill the condition of Proximity Criteria.
 * @param D Max distance to check.
 * @param tCount Number of t values.
 * @param tValues Array of t values.
 * @param points Array of points.
 * @param results Array of results.
 * @return 0 if the computation is successful.
 */
int computeOnGPU(int N, int K, double D, int tCount, double *tValues, Point *points, int *results)
{
    hipError_t err = hipSuccess;

    int threadPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (N + threadPerBlock - 1) / threadPerBlock;
    Point *d_points = NULL;
    int *d_results = NULL;

    /*Allocating mem on gpu section*/
    allocateMemDevice((void **)&d_results, CONSTRAINTS * tCount * sizeof(int)); 
    allocateMemDevice((void **)&d_points, N * sizeof(Point));
    /*End allocate mem*/

    /*Copy mem to Device section*/
    copyMemory(d_points, points, N * sizeof(Point), hipMemcpyHostToDevice);
    copyMemory(d_results, results, tCount * CONSTRAINTS * sizeof(int), hipMemcpyHostToDevice);
    /*End copy mem to Device*/

    /*for each tvalue we will send it to GPU to compute the data and save it on results array*/
    for (int i = 0; i < tCount; i++)
    {
        checkProximity<<<blocksPerGrid, threadPerBlock>>>(d_points, N, tValues[i], D, d_results, K, i);
        err = hipGetLastError();
        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to lanch checkProximity kernel. -%s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }

    err = hipMemcpy(results, d_results, tCount * CONSTRAINTS * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data. -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data. -%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (hipFree(d_points) != hipSuccess)
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if (hipFree(d_results) != hipSuccess)
    {
        fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return 0;
}
